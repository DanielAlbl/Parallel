#include "hip/hip_runtime.h"
/*
 * Daniel Albl
 * CSC 410 2020
 * Exam 1
 *
 * This program implements and tests a parallel version of Floyd's algorithm using CUDA 
 */

#include "global.h"

const int BLOCK_SIZE = 256;
int GRID_SIZE, SIZE;
int* _A;

__global__  
void floyd(int* _A, int N) {
	int tmp, idx = blockIdx.x*blockDim.x + threadIdx.x;
	for(int k = 0; k < N; k++) {
		tmp = _A[N*(idx/N) + k] + _A[N*k + (idx%N)];
		if(tmp < _A[idx] and tmp > 0)
			_A[idx] = tmp;
		__syncthreads();
	}
}

bool test() {
	bool passed;
	A = new int[100];
	hipMalloc((void**)&_A, 100 * sizeof(int));

	hipMemcpy(_A, t11, 36 * sizeof(int), hipMemcpyHostToDevice);
	floyd<<<1, 36>>>(_A, 6);
	hipMemcpy(A, _A, 36 * sizeof(int), hipMemcpyDeviceToHost);
	passed = !memcmp(t12, A, 36 * sizeof(int));

	hipMemcpy(_A, t21, 100 * sizeof(int), hipMemcpyHostToDevice);
	floyd<<<1, 100>>>(_A, 10);
	hipMemcpy(A, _A, 100 * sizeof(int), hipMemcpyDeviceToHost);
	passed = passed and !memcmp(t22, A, 100 * sizeof(int));

	free();
	hipFree(_A);
	
	return passed;
}

int main(int argc, char** argv) {
	N = stoi(argv[1]);
	init();

	SIZE = N*N*sizeof(int);
	GRID_SIZE = N*N / BLOCK_SIZE + (BLOCK_SIZE % N*N) ? 0 : 1;

	// Since you have to copy to gpu in cuda and not omp
	// I think it's fair to include it in the timing
	double start = omp_get_wtime();
	hipMalloc((void**)&_A, SIZE);
	hipMemcpy(_A, A, SIZE, hipMemcpyHostToDevice);
	floyd<<<GRID_SIZE, BLOCK_SIZE>>>(_A, N);
	double end = omp_get_wtime();

	cout << "Time elapsed: " << end - start << "s\n";
	free();
	hipFree(_A);
}
