/* Introduction code to CUDA
 * Second attempt to push this onto the device
 * 
 * Compile: nvcc -g -o vec_add vecAdd2.cu -lm
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int index = threadIdx.x;
    int stride = blockDim.x;
 
    // Make sure we do not go out of bounds
    for (int i = index; i<n; i += stride)
        c[i] = a[i] + b[i];
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 1<<20;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    // Number of threads in each thread block
    int blockSize = 256;
 
    // Execute the kernel
    vecAdd<<<1, blockSize>>>(d_a, d_b, d_c, n);
 
   // Wait for the GPU to finish
   hipDeviceSynchronize(); 

   // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
